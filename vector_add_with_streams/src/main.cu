#include "hip/hip_runtime.h"
﻿// CUDA runtime
#include <hip/hip_runtime.h>
#include <>

// Own includes
#include "common/helpers.h"

constexpr int numElements = 1'000'000;

inline float calcKahamSum(const float* resultVec) {
    float sum = 0.0f;
    float err = 0.0f;
    for (int i = 0; i < numElements; i++) {
        float currVal = resultVec[i] - err;
        float currSum = sum + currVal;
        err = (currSum - sum) - currVal;
        sum = currSum;
    }
    return sum;
}

inline void cpuVectorAdd(float* hostA, float* hostB, float* hostC) {
    for (int i = 0; i < numElements; i++) {
        hostC[i] = hostA[i] + hostB[i];
    }
}

// Compute vector sum C = A + B
__global__ void vectorAddKernel(const float* A, const float* B, float* C, const int n) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        C[tid] = A[tid] + B[tid];
    }
}

void launchGPUVectorAdd(const float* hostA, const float* hostB) {
    // alloc peagable host memory for the result
    const int numBytes = numElements * sizeof(float);
    float* hostC = (float*)malloc(numBytes);

    // create cuda event handles to measure end-to-end and kernel time
    hipEvent_t start, stop, kernelStart, kernelStop;
    handleCUDAError(hipEventCreate(&start));
    handleCUDAError(hipEventCreate(&stop));
    handleCUDAError(hipEventCreate(&kernelStart));
    handleCUDAError(hipEventCreate(&kernelStop));

    // start to measure end-to-end time execution
    handleCUDAError(hipEventRecord(start, 0));

    // allocate memory on the device
    float *devA, *devB, *devC;
    handleCUDAError(hipMalloc((void**)&devA, numBytes));
    handleCUDAError(hipMalloc((void**)&devB, numBytes));
    handleCUDAError(hipMalloc((void**)&devC, numBytes));

    // transfer host data to the device
    handleCUDAError(hipMemcpy(devA, hostA, numBytes, hipMemcpyHostToDevice));
    handleCUDAError(hipMemcpy(devB, hostB, numBytes, hipMemcpyHostToDevice));

    // kernel execution config
    const int blockSize = 256;
    const int gridSize = (numElements + blockSize - 1) / blockSize;

    // run the kernele and measure execution time
    handleCUDAError(hipEventRecord(kernelStart, 0));
    vectorAddKernel<<<gridSize, blockSize>>>(devA, devB, devC, numElements);
    handleCUDAError(hipEventRecord(kernelStop, 0));

    // synchronize with kernel execution
    handleCUDAError(hipEventSynchronize(kernelStop));

    // thansfer data back to the host
    handleCUDAError(hipMemcpy(hostC, devC, numBytes, hipMemcpyDeviceToHost));

    // stop to measure end-to-end time
    handleCUDAError(hipEventRecord(stop, 0));

    // synchronize with data stransfer
    handleCUDAError(hipEventSynchronize(stop));

    float kernelTime, endToEndTime;
    handleCUDAError(hipEventElapsedTime(&kernelTime, kernelStart, kernelStop));
    handleCUDAError(hipEventElapsedTime(&endToEndTime, start, stop));

    const float devResult = calcKahamSum(hostC);
    fprintf(stdout,
            "GPU time for vector addition of %d elements: [%.4f] - result %f, end-to-end time: "
            "[%.4f]; executed on default stream\n",
            numElements, kernelTime, devResult, endToEndTime);

    // destroy events
    handleCUDAError(hipEventDestroy(kernelStart));
    handleCUDAError(hipEventDestroy(kernelStop));
    handleCUDAError(hipEventDestroy(start));
    handleCUDAError(hipEventDestroy(stop));

    // free device memory
    handleCUDAError(hipFree(devA));
    handleCUDAError(hipFree(devB));
    handleCUDAError(hipFree(devC));
}

void launchGPUVectorAddAsync(const float* hostA, const float* hostB) {
    // alloc pinned host memory for the result
    const int numBytes = numElements * sizeof(float);
    float* hostC;
    handleCUDAError(hipHostMalloc((void**)&hostC, numBytes));

    // allocate memory on the device
    float *devA, *devB, *devC;
    handleCUDAError(hipMalloc((void**)&devA, numBytes));
    handleCUDAError(hipMalloc((void**)&devB, numBytes));
    handleCUDAError(hipMalloc((void**)&devC, numBytes));

    // create streams
    constexpr int numStreams = 10;
    constexpr int streamSize = numElements / numStreams;
    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; i++) {
        handleCUDAError(hipStreamCreate(&streams[i]));
    }

    // create cuda event handles to measure end-to-end and kernel time
    hipEvent_t start, stop, kernelStart, kernelStop;
    handleCUDAError(hipEventCreate(&start));
    handleCUDAError(hipEventCreate(&stop));
    handleCUDAError(hipEventCreate(&kernelStart));
    handleCUDAError(hipEventCreate(&kernelStop));

    // kernel execution config
    constexpr int blockSize = 256;
    constexpr int gridSize = (numElements + blockSize - 1) / blockSize;

    // start to measure end-to-end time execution
    handleCUDAError(hipEventRecord(start, 0));

    // thansfer data to the device asynchronously
    for (int i = 0; i < numStreams; i++) {
        int offset = streamSize * i;
        handleCUDAError(hipMemcpyAsync(&devA[offset], &hostA[offset], streamSize * sizeof(float),
                                        hipMemcpyHostToDevice, streams[i]));
        handleCUDAError(hipMemcpyAsync(&devB[offset], &hostB[offset], streamSize * sizeof(float),
                                        hipMemcpyHostToDevice, streams[i]));
    }

    // start to measure kernel time
    handleCUDAError(hipEventRecord(kernelStart, 0));
    for (int i = 0; i < numStreams; i++) {
        int offset = streamSize * i;
        vectorAddKernel<<<(gridSize + numStreams - 1) / numStreams, blockSize, 0, streams[i]>>>(
            &devA[offset], &devB[offset], &devC[offset], streamSize);
    }
    // stop to measure kernel time
    handleCUDAError(hipEventRecord(kernelStop, 0));

    // thansfer data back to the host
    for (int i = 0; i < numStreams; i++) {
        int offset = streamSize * i;
        handleCUDAError(hipMemcpyAsync(&hostC[offset], &devC[offset], streamSize * sizeof(float),
                                        hipMemcpyDeviceToHost, streams[i]));
    }

    // stop to measure end-to-end time
    handleCUDAError(hipEventRecord(stop, 0));

    // synchronize with data transfer
    handleCUDAError(hipEventSynchronize(stop));

    float kernelTime, endToEndTime;
    handleCUDAError(hipEventElapsedTime(&kernelTime, kernelStart, kernelStop));
    handleCUDAError(hipEventElapsedTime(&endToEndTime, start, stop));

    const float devResult = calcKahamSum(hostC);
    fprintf(stdout,
            "GPU time for vector addition of %d elements: [%.4f] - result %f, end-to-end time: "
            "[%.4f]; executed on %d streams\n",
            numElements, kernelTime, devResult, endToEndTime, numStreams);

    // destroy events
    handleCUDAError(hipEventDestroy(kernelStart));
    handleCUDAError(hipEventDestroy(kernelStop));
    handleCUDAError(hipEventDestroy(start));
    handleCUDAError(hipEventDestroy(stop));

    // destroy streams
    for (int i = 0; i < numStreams; i++) {
        handleCUDAError(hipStreamDestroy(streams[i]));
    }

    // free device memory
    handleCUDAError(hipFree(devA));
    handleCUDAError(hipFree(devB));
    handleCUDAError(hipFree(devC));

    // free pinned host memory
    handleCUDAError(hipHostFree(hostC));
}

int main() {
    using nanosec = std::chrono::nanoseconds;
    using high_res_clock = std::chrono::high_resolution_clock;

    // allocate memmory on the host
    float* hostA = (float*)malloc(numElements * sizeof(float));
    float* hostB = (float*)malloc(numElements * sizeof(float));
    float* hostC = (float*)malloc(numElements * sizeof(float));

    // generate random numbers
    for (int i = 0; i < numElements; i++) {
        hostA[i] = rand() % numElements;
        hostB[i] = rand() % numElements;
    }

    // measure host time
    {
        auto hostStartTime = high_res_clock::now();
        cpuVectorAdd(hostA, hostB, hostC);
        auto hostEndTime = high_res_clock::now();

        const float hostResult = calcKahamSum(hostC);
        const float cpuTime = cpuDuration<nanosec>(hostStartTime, hostEndTime) / 1e6;
        fprintf(stdout, "CPU time for vector addition of %d elements: [%.4f] - result %f\n",
                numElements, cpuTime, hostResult);
    }

    // run on defalut stream and measure time
    launchGPUVectorAdd(hostA, hostB);

    // run async on non-default streams and measure time
    launchGPUVectorAddAsync(hostA, hostB);

    // free host memory
    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
